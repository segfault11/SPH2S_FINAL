#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
//  Solver.cu
//------------------------------------------------------------------------------
#include "Solver.h"
//------------------------------------------------------------------------------

//==============================================================================
//  CUDA DEVICE code starts here 
//==============================================================================

//------------------------------------------------------------------------------
__constant__ SolverConfiguration gConfiguration;    // current solver's config.
//------------------------------------------------------------------------------
enum 
{
    LOW_RES = 0,
    HIGH_RES = 1
};
//------------------------------------------------------------------------------
#define COMPL(x) (x + 1) % 2
//------------------------------------------------------------------------------

//==============================================================================
// UTLITY device kernels definition
//==============================================================================

//------------------------------------------------------------------------------
__device__ void computeCoordinatesOff(
    int3& coordinate,            // out: coordinate for [position]
    float3 position,
    const Grid& grid,
    float offset
)
{
    // compute the coordinates of a point in space with respect to the given 
    // grid

    coordinate.x = (int)((position.x + offset - grid.Origin.x)/grid.Spacing);
    coordinate.y = (int)((position.y + offset - grid.Origin.y)/grid.Spacing);
    coordinate.z = (int)((position.z + offset - grid.Origin.z)/grid.Spacing);

    // clamp coordinates if neccessary
    coordinate.x = max(0, min(coordinate.x, grid.Dimensions.x - 1));
    coordinate.y = max(0, min(coordinate.y, grid.Dimensions.y - 1));
    coordinate.z = max(0, min(coordinate.z, grid.Dimensions.z - 1));
}
//------------------------------------------------------------------------------
__device__ void computeCoordinates(
    int3& coordinate,            // out: coordinate for [position]
    float3 position,
    const Grid& grid
)
{
    // compute the coordinates of a point in space with respect to the given 
    // grid

    coordinate.x = (int)((position.x - grid.Origin.x)/grid.Spacing);
    coordinate.y = (int)((position.y - grid.Origin.y)/grid.Spacing);
    coordinate.z = (int)((position.z - grid.Origin.z)/grid.Spacing);

    // clamp coordinates if neccessary
    coordinate.x = max(0, min(coordinate.x, grid.Dimensions.x - 1));
    coordinate.y = max(0, min(coordinate.y, grid.Dimensions.y - 1));
    coordinate.z = max(0, min(coordinate.z, grid.Dimensions.z - 1));
}
//------------------------------------------------------------------------------
__device__ void computeHash(
    unsigned int& hash, 
    const int3& coordinate,
    const Grid& grid
)
{
    // compute the hash for a grid given a coordinate within the grid

    hash = coordinate.x + grid.Dimensions.x*
        (coordinate.y + grid.Dimensions.y*coordinate.z);
}
//------------------------------------------------------------------------------
__device__ void computeHash(
    unsigned int& hash, 
    float3 position,
    const Grid& grid
)
{
    // compute the hash for a grid given a position in world space, by first
    // conputing the coordinate in [grid], and then computing the hash.

    int3 coordinate;
    computeCoordinates(coordinate, position, grid);
    computeHash(hash, coordinate, grid);
}
//------------------------------------------------------------------------------
__device__ inline void computeNorm(float& norm, const float3& a)
{
    norm = sqrt(a.x*a.x + a.y*a.y + a.z*a.z);
}
//------------------------------------------------------------------------------
__device__ inline void computeDistance(
    float& dist, 
    const float3& a, 
    const float3& b
)
{
    float3 d;
    d.x = a.x - b.x;
    d.y = a.y - b.y;
    d.z = a.z - b.z;
    computeNorm(dist, d); 
}
//------------------------------------------------------------------------------
__device__ inline void evaluatePoly6Kernel(
    float& res,  // [out] result of evaluation
    float d,     // distance between two particles
    float h      // effective radius 
)
{
    // evaluate Muellers Poly6 Kernel

    float hhh = h*h*h;
    float coeff = 315.0f/(64.0f*M_PI*hhh*hhh*hhh);

    if (d < h)
    {
        float a = h*h - d*d;
        res = coeff*a*a*a;
    }
    else
    {
        res = 0.0f;    
    }
}
//------------------------------------------------------------------------------
__device__ inline void evaluateSpikyKernelGradient(
    float3& grad,
    const float3& xij,
    float h
)
{
    float norm = 0.0f;
    computeNorm(norm, xij);
    
    if ((norm == 0.0f) || (norm > h))
    {
        grad.x = 0.0f;
        grad.y = 0.0f;
        grad.z = 0.0f;
        return;
    } 

    float hhh = h*h*h;
    float a = -45.0f/(M_PI*hhh*hhh)*(h - norm)*(h - norm);

    grad.x = a*xij.x/norm;
    grad.y = a*xij.y/norm;
    grad.z = a*xij.z/norm;
}
//------------------------------------------------------------------------------
__device__ inline void evaluateViscosityKernelLaplacian(
    float& lapl,
    float dist,
    float h
)
{
    if (dist < h)
    {
        float hhh = h*h*h;
        float coeff = 45.0f/(M_PI*hhh*hhh);
        lapl = coeff*(h - dist);
        return;
    }
    else
    {
        return;
    }
}
//------------------------------------------------------------------------------
__device__ inline void evaluateBoundaryWeight(
    float& weight, 
    float dist, 
    float h
)
{
    float q = 2.0f*dist/h;
    float coeff = 0.02f*gConfiguration.SpeedSound*gConfiguration.SpeedSound/
        dist;

    if (q < 2.0f/3.0f)
    {
        weight = coeff*2.0f/3.0f;
    } 
    else if (q < 1.0f)
    {
        weight = coeff*(2.0f*q - 3.0f/2.0f*q*q);
    }
    else if (q < 2.0f)
    {
        float a = 2.0f - q;
        weight = coeff*0.5f*a*a;
    }
    else
    {
        weight = 0.0f;
    }
}
//------------------------------------------------------------------------------
#define MEXICAN_HAT_C 0.8673250705840776f // c =  2/(sqrt(3)*pi^(1/4))
__device__ inline float evaluateMexicanHat3D(float x, float y, float z)
{
	x = x*x;
	y = y*y;
	z = z*z;
	return MEXICAN_HAT_C*(x+y+z-3.0f)*exp(-(x+y+z)/2.0f);
}
//------------------------------------------------------------------------------
__device__ inline void computeDensityCell(
    float& rhoi,                 // [out] density of particle i 
    const float3& xi,            // position of particle i
    const float* dPositions,     
    const float* dBlendCoefficients,
    unsigned int start,
    unsigned int end,
    int resID
)
{
    // add up density contribution form particle in this cell ([start], [end])
    // to the density of the particle i [rhoi]. (in fact only the kernel 
    // weights are added up, mass is multiplied in the callee, to safe
    // operations)
    
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float lambdaj = dBlendCoefficients[j];
        float dist;
        computeDistance(dist, xi, xj);

        if (dist < gConfiguration.EffectiveRadius[resID])
        {
            float weight = 0.0f;
            evaluatePoly6Kernel(weight, dist, 
                gConfiguration.EffectiveRadius[resID]);
            rhoi += lambdaj*weight;
        }
            
    }
  
}
//------------------------------------------------------------------------------
__device__ inline void computeDensityCellCompl(
    float& rhoi,                 // [out] density of particle i 
    const float3& xi,            // position of particle i
    const float* dPositions,    
    const float* dBlendCoefficients, 
    unsigned int start,
    unsigned int end
)
{
    // add up density contribution form particle in this cell ([start], [end])
    // to the density of the particle i [rhoi]. (in fact only the kernel 
    // weights are added up, mass is multiplied in the callee, to safe
    // operations)
    
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float lambdaj = dBlendCoefficients[j];
        float dist;
        computeDistance(dist, xi, xj);

        if (dist < gConfiguration.EffectiveRadius[LOW_RES])
        {
            float weight0 = 0.0f;
            float weight1 = 0.0f;

            evaluatePoly6Kernel(weight0, dist, 
                gConfiguration.EffectiveRadius[LOW_RES]);
            evaluatePoly6Kernel(weight1, dist, 
                gConfiguration.EffectiveRadius[HIGH_RES]);
            rhoi += lambdaj*0.5f*(weight0 + weight1);
        }
            
    }
  
}
//------------------------------------------------------------------------------
__device__ inline void computeAccelerationCell(
    float3& fi,
    float3& velW,
    float& psiSum,
    float3& xc,
    float& massSum,
    float rhoi,
    float pi,
    const float3& xi,        
    const float3& vi,    
    const float* dDensities,
    const float* dPressures,
    const float* dPositions,     
    const float* dVelocities,
    const float* dBlendCoefficients,
    unsigned int start,
    unsigned int end,
    int resID
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float dist;
        float3 xij;
        xij.x = xi.x - xj.x; 
        xij.y = xi.y - xj.y; 
        xij.z = xi.z - xj.z; 
        computeNorm(dist, xij);
        
        if (dist != 0.0f && dist < gConfiguration.EffectiveRadius[resID])
        {
            float3 vj;
            vj.x = dVelocities[3*j + 0];
            vj.y = dVelocities[3*j + 1];
            vj.z = dVelocities[3*j + 2];
            float rhoj = dDensities[j];
            float pj = dPressures[j];
            float lambdaj = dBlendCoefficients[j];
            float mj = gConfiguration.FluidParticleMass[resID];

            // evaluate the pressure force partice j exerts on particle i
            float coeffP = -rhoi*mj*(pi/(rhoi*rhoi) + pj/(rhoj*rhoj));
            float3 grad;
            evaluateSpikyKernelGradient(
                grad, 
                xij,
                gConfiguration.EffectiveRadius[resID]
            );
            fi.x += lambdaj*coeffP*grad.x;
            fi.y += lambdaj*coeffP*grad.y;
            fi.z += lambdaj*coeffP*grad.z;

            // evaluate the viscosity force partice j exerts on particle i
            float coeffV = gConfiguration.Viscosity*mj/rhoj;
            float lapl = 0.0f;
            evaluateViscosityKernelLaplacian(
                lapl, 
                dist, 
                gConfiguration.EffectiveRadius[resID]
            );
            float3 vji;
            vji.x = vj.x - vi.x;
            vji.y = vj.y - vi.y;
            vji.z = vj.z - vi.z;
            fi.x += lambdaj*coeffV*vji.x*lapl;
            fi.y += lambdaj*coeffV*vji.y*lapl;
            fi.z += lambdaj*coeffV*vji.z*lapl;

            // evaluate the surface tension force partice j exerts on particle i
            float weight;
            evaluatePoly6Kernel(
                weight, 
                dist, 
                gConfiguration.EffectiveRadius[resID]
            );
            float coeffT = -weight*mj*gConfiguration.TensionCoefficient;
        
            fi.x += lambdaj*coeffT*xij.x;
            fi.y += lambdaj*coeffT*xij.y;
            fi.z += lambdaj*coeffT*xij.z;

            float h = gConfiguration.EffectiveRadius[resID];
            float psi = evaluateMexicanHat3D(xij.x/h, xij.y/h, xij.z/h);
            velW.x += vj.x*psi;
            velW.y += vj.y*psi;
            velW.z += vj.z*psi;
            psiSum += psi;

            xc.x += mj*xj.x;
            xc.y += mj*xj.y;
            xc.z += mj*xj.z;
            massSum += mj;
        }

    }

}
//------------------------------------------------------------------------------
__device__ inline void computeAccelerationCellCompl(
    float3& fi,
    float rhoi,
    float pi,
    const float3& xi,        
    const float3& vi,    
    const float* dDensities,
    const float* dPressures,
    const float* dPositions,     
    const float* dVelocities,
    const float* dBlendCoefficients,
    unsigned int start,
    unsigned int end,
    int resID         // resolution id for complement domain
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float3 vj;
        vj.x = dVelocities[3*j + 0];
        vj.y = dVelocities[3*j + 1];
        vj.z = dVelocities[3*j + 2];
        float rhoj = dDensities[j];
        float pj = dPressures[j];
        float lambdaj = dBlendCoefficients[j];
        float dist;
        float3 xij;
        xij.x = xi.x - xj.x; 
        xij.y = xi.y - xj.y; 
        xij.z = xi.z - xj.z; 
        computeNorm(dist, xij);
        
        if (dist != 0.0f && dist < gConfiguration.EffectiveRadius[LOW_RES])
        {
            // evaluate the pressure force partice j exerts on particle i
            float coeffP = -rhoi*gConfiguration.FluidParticleMass[resID]*
                (pi/(rhoi*rhoi) + pj/(rhoj*rhoj));
            float3 grad0;
            float3 grad1;
            evaluateSpikyKernelGradient(
                grad0, 
                xij,
                gConfiguration.EffectiveRadius[LOW_RES]
            );
            evaluateSpikyKernelGradient(
                grad1, 
                xij,
                gConfiguration.EffectiveRadius[HIGH_RES]
            );
            fi.x += lambdaj*coeffP*0.5*(grad0.x + grad1.x);
            fi.y += lambdaj*coeffP*0.5*(grad0.y + grad1.y);
            fi.z += lambdaj*coeffP*0.5*(grad0.z + grad1.z);

            // evaluate the viscosity force partice j exerts on particle i
            float coeffV = gConfiguration.Viscosity*
                gConfiguration.FluidParticleMass[resID]/rhoj;
            float lapl0 = 0.0f;
            float lapl1 = 0.0f;
            evaluateViscosityKernelLaplacian(
                lapl0, 
                dist, 
                gConfiguration.EffectiveRadius[LOW_RES]
            );
            evaluateViscosityKernelLaplacian(
                lapl1, 
                dist, 
                gConfiguration.EffectiveRadius[HIGH_RES]
            );
            float3 vji;
            vji.x = vj.x - vi.x;
            vji.y = vj.y - vi.y;
            vji.z = vj.z - vi.z;
            float lapl = 0.5f*(lapl0 + lapl1);
            fi.x += lambdaj*coeffV*vji.x*lapl;
            fi.y += lambdaj*coeffV*vji.y*lapl;
            fi.z += lambdaj*coeffV*vji.z*lapl;

            // evaluate the surface tension force partice j exerts on particle i
            float weight0;
            float weight1;
            evaluatePoly6Kernel(
                weight0, 
                dist, 
                gConfiguration.EffectiveRadius[LOW_RES]
            );
            evaluatePoly6Kernel(
                weight1, 
                dist, 
                gConfiguration.EffectiveRadius[HIGH_RES]
            );
            float weight = 0.5f*(weight0 + weight1);
            float coeffT = -weight*gConfiguration.FluidParticleMass[resID]*
                gConfiguration.TensionCoefficient;
        
            fi.x += lambdaj*coeffT*xij.x;
            fi.y += lambdaj*coeffT*xij.y;
            fi.z += lambdaj*coeffT*xij.z;
        }

    }

}
//------------------------------------------------------------------------------
__device__ void computeBoundaryForceCell(
    float3& bi,
    const float3& xi,
    const float* dPositions,     
    unsigned int start,
    unsigned int end,
    int resID
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float3 xij;
        xij.x = xi.x - xj.x;
        xij.y = xi.y - xj.y;
        xij.z = xi.z - xj.z;
        float dist;
        computeNorm(dist, xij); 

        if (dist < gConfiguration.EffectiveRadius[resID])
        {
            float weight = 0.0f;
            evaluateBoundaryWeight(
                weight, 
                dist, 
                gConfiguration.EffectiveRadius[resID]
            );
            weight*= gConfiguration.BoundaryParticleMass/
                (gConfiguration.FluidParticleMass[resID] + 
                gConfiguration.BoundaryParticleMass);
            bi.x += weight*xij.x/dist;
            bi.y += weight*xij.y/dist;
            bi.z += weight*xij.z/dist;
        }
    }
}
//------------------------------------------------------------------------------
#define SQRT3INV 0.57735026919
__device__ void insertHighResParticles(
    float* dPositions,              // positions of the high res particles
    float* dVelocities,             // velocities of the high res particles
    float* dBlendCoefficients,      // blend vals of the high res particles
    int* dStates,         // states of the high res particles
    unsigned int* dActiveIDs,       // list of active high particle ids
    unsigned int* dNumParticles,    // # particles in the list
    const float3& posLow,           // position of the particle that is split
    const float3& velLow,           // velocity of the particle that is split
    float densLow                   // density of the particle that is split
)
{
    const float dir[] = {
         SQRT3INV,  SQRT3INV,  SQRT3INV,
         SQRT3INV,  SQRT3INV, -SQRT3INV,
         SQRT3INV, -SQRT3INV,  SQRT3INV,
         SQRT3INV, -SQRT3INV, -SQRT3INV,
        -SQRT3INV,  SQRT3INV,  SQRT3INV,
        -SQRT3INV,  SQRT3INV, -SQRT3INV,
        -SQRT3INV, -SQRT3INV,  SQRT3INV,
        -SQRT3INV, -SQRT3INV, -SQRT3INV
    };
    
    // add eight to the high res particles
    unsigned int id = atomicAdd(dNumParticles, 8);
    float r = 0.5f*pow(
            3.0f/(4.0f*float(M_PI))*
            gConfiguration.FluidParticleMass[LOW_RES]/densLow,
            1.0f/3.0f
        );

    for (unsigned int i = 0; i < 8; i++)
    {
        dActiveIDs[id + i] = id + i;
        dBlendCoefficients[id + i] = 0.0f;
        dStates[id + i] = 0x06;
        dPositions[3*(id + i) + 0] = posLow.x + r*dir[3*i + 0];
        dPositions[3*(id + i) + 1] = posLow.y + r*dir[3*i + 1];
        dPositions[3*(id + i) + 2] = posLow.z + r*dir[3*i + 2];
        dVelocities[3*(id + i) + 0] = velLow.x;
        dVelocities[3*(id + i) + 1] = velLow.y;
        dVelocities[3*(id + i) + 2] = velLow.z;
    }

}
//------------------------------------------------------------------------------

//==============================================================================
// GLOBAL device kernel definitions
//==============================================================================

//------------------------------------------------------------------------------
__global__ void computeHashs(
    unsigned int* dHashs,           // hash values of each particle
    unsigned int* dActiveIDs,       // array of active particle ids
    const float* dPositions,        // positions of each particle 
    unsigned int numParticles,      // number of ids in the id array
    int resID
)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    unsigned int id = dActiveIDs[idx];

    float3 pos;
    pos.x = dPositions[3*id + 0];
    pos.y = dPositions[3*id + 1];
    pos.z = dPositions[3*id + 2];

    computeHash(dHashs[idx], pos, gConfiguration.Grid[resID]);
};
//------------------------------------------------------------------------------
__global__ void reorderComputeCellStartEndBoundaryD(
    unsigned int* dCellStart,
    unsigned int* dCellEnd,
    float* dTempPositions,
    const float* dPositions,
    const unsigned int* dSortedIDs,
    const unsigned int* dHashs,
    unsigned int numParticles
)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles) 
    {
        return;
    }

    // reorder
    unsigned int id = dSortedIDs[idx];
    dTempPositions[3*idx + 0] = dPositions[3*id + 0];
    dTempPositions[3*idx + 1] = dPositions[3*id + 1];
    dTempPositions[3*idx + 2] = dPositions[3*id + 2];

    // compute cell start end
    int hash = dHashs[idx];
    sharedHash[threadIdx.x + 1] = hash;
        
    if (idx > 0 && threadIdx.x == 0) 
    {
        sharedHash[0] = dHashs[idx - 1];
    }

    __syncthreads();

    if (idx == 0 || hash != sharedHash[threadIdx.x])
    {
        dCellStart[hash] = idx;
        
        if (idx > 0) 
        {
            dCellEnd[sharedHash[threadIdx.x]] = idx;
        }
    }

    if (idx == numParticles - 1)
    {
        dCellEnd[hash] = idx + 1;
    }
}
//------------------------------------------------------------------------------
__global__ void reorderAndComputeCellStartEndD(
    unsigned int* dCellStart,
    unsigned int* dCellEnd,
    float* dTempPositions,
    float* dTempVelocities,
    int* dTempStates,
    float* dTempBlendCoefficients,
    const unsigned int* dSortedIDs,
    const float* dPositions,
    const float* dVelocities,
    const int* dStates,
    const float* dBlendCoefficients,
    const unsigned int* dHashs,
    unsigned int numParticles
)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles) 
    {
        return;
    }
    
    // reorder
    unsigned int id = dSortedIDs[idx];
    dTempPositions[3*idx + 0] = dPositions[3*id + 0]; 
    dTempPositions[3*idx + 1] = dPositions[3*id + 1]; 
    dTempPositions[3*idx + 2] = dPositions[3*id + 2]; 
    dTempVelocities[3*idx + 0] = dVelocities[3*id + 0]; 
    dTempVelocities[3*idx + 1] = dVelocities[3*id + 1]; 
    dTempVelocities[3*idx + 2] = dVelocities[3*id + 2];
    dTempStates[idx] = dStates[id];
    dTempBlendCoefficients[idx] = dBlendCoefficients[id];
    
    // compute cell start end ids
    int hash = dHashs[idx];
    sharedHash[threadIdx.x + 1] = hash;
        
    if (idx > 0 && threadIdx.x == 0) 
    {
        sharedHash[0] = dHashs[idx - 1];
    }

    __syncthreads();

    if (idx == 0 || hash != sharedHash[threadIdx.x])
    {
        dCellStart[hash] = idx;
        
        if (idx > 0) 
        {
            dCellEnd[sharedHash[threadIdx.x]] = idx;
        }
    }

    if (idx == numParticles - 1)
    {
        dCellEnd[hash] = idx + 1;
    }
}
//------------------------------------------------------------------------------
__global__ void computeDensitiesPressuresD(
    float* dDensities,              // [out] computed densities
    float* dPressures,
    const float* dPositions,
    const float* dBlendCoefficients,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const float* dPositionsCompl,
    const float* dBlendCoefficientsCompl,
    const unsigned int* dCellStartCompl,
    const unsigned int* dCellEndCompl,
    unsigned int numParticles,
    int resID             // resolution identifier
)
{
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];
    float rhoi = 0.0f;
    float rhoiCompl = 0.0f;
    int3 cs, ce, cc;

    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[resID], 
        -gConfiguration.EffectiveRadius[resID]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[resID], 
        gConfiguration.EffectiveRadius[resID]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[resID]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeDensityCell(
                    rhoi,
                    xi,
                    dPositions,
                    dBlendCoefficients,
                    start,
                    end,
                    resID
                );
            }
        }
    }

    //--------------------------------------------------------------------------
    // compute density contribution of the complementary domain
    //--------------------------------------------------------------------------

    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[COMPL(resID)], 
        -gConfiguration.EffectiveRadius[LOW_RES]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[COMPL(resID)], 
        gConfiguration.EffectiveRadius[LOW_RES]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[COMPL(resID)]);
                unsigned int start = dCellStartCompl[hash];
                unsigned int end = dCellEndCompl[hash];

                computeDensityCellCompl(
                    rhoiCompl,
                    xi,
                    dPositionsCompl,
                    dBlendCoefficientsCompl,
                    start,
                    end
                );
            }
        }
    }


    rhoi = rhoi*gConfiguration.FluidParticleMass[resID] + 
        rhoiCompl*gConfiguration.FluidParticleMass[COMPL(resID)];
    dDensities[idx] = rhoi;
    dPressures[idx] = gConfiguration.BulkModulus*
        (rhoi - gConfiguration.RestDensity);
}
//------------------------------------------------------------------------------
__global__ void computeAccelerationsD(
    float* dAccelerations,
    float* dColorValues,
    const float* dDensities,              
    const float* dPressures,
    const float* dPositions,
    const float* dVelocities,
    const float* dBlendCoefficients,
    int* dStates,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const float* dDensitiesCompl,              
    const float* dPressuresCompl,
    const float* dPositionsCompl,
    const float* dVelocitiesCompl,
    const float* dBlendCoefficientsCompl,
    const unsigned int* dCellStartCompl,
    const unsigned int* dCellEndCompl,
    const float* dBoundaryPositions,
    const unsigned int* dBoundaryCellStart,
    const unsigned int* dBoundaryCellEnd,
    unsigned int numParticles,
    int resID
)
{
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];
    float3 vi;
    vi.x = dVelocities[3*idx + 0];
    vi.y = dVelocities[3*idx + 1];
    vi.z = dVelocities[3*idx + 2];
    float rhoi = dDensities[idx];
    float pi = dPressures[idx];
    float3 fi;
    fi.x = 0.0;
    fi.y = 0.0;
    fi.z = 0.0;
    float3 bi;
    bi.x = 0.0f;
    bi.y = 0.0f;
    bi.z = 0.0f;
    float psi = evaluateMexicanHat3D(0.0f, 0.0f, 0.0f);
    float psiSum = psi;
    float3 velW;
    velW.x = vi.x*psi;
    velW.y = vi.y*psi;
    velW.z = vi.z*psi;
    float massSum = gConfiguration.FluidParticleMass[resID];
    float3 xc;
    xc.x = gConfiguration.FluidParticleMass[resID]*xi.x;
    xc.y = gConfiguration.FluidParticleMass[resID]*xi.y; 
    xc.z = gConfiguration.FluidParticleMass[resID]*xi.z;
    int3 cc, cs, ce;

    //--------------------------------------------------------------------------
    // compute force contribution of the same domain
    //--------------------------------------------------------------------------
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[resID], 
        -gConfiguration.EffectiveRadius[resID]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[resID], 
        gConfiguration.EffectiveRadius[resID]
    );
    
    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[resID]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeAccelerationCell(
                    fi,
                    velW,
                    psiSum,
                    xc,
                    massSum,
                    rhoi,
                    pi,
                    xi,
                    vi,
                    dDensities,
                    dPressures,
                    dPositions,
                    dVelocities,
                    dBlendCoefficients,
                    start,
                    end,
                    resID
                );
            }
        }
    }

    //--------------------------------------------------------------------------
    // compute force contribution of the complementary domain
    //--------------------------------------------------------------------------
    
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[COMPL(resID)], 
        -gConfiguration.EffectiveRadius[LOW_RES]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[COMPL(resID)], 
        gConfiguration.EffectiveRadius[LOW_RES]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[COMPL(resID)]);
                unsigned int start = dCellStartCompl[hash];
                unsigned int end = dCellEndCompl[hash];

                computeAccelerationCellCompl(
                    fi,
                    rhoi,
                    pi,
                    xi,
                    vi,
                    dDensitiesCompl,
                    dPressuresCompl,
                    dPositionsCompl,
                    dVelocitiesCompl,
                    dBlendCoefficientsCompl,
                    start,
                    end,
                    COMPL(resID)
                );
            }
        }
    }

    //--------------------------------------------------------------------------
    // compute force contribution of the boundary
    //--------------------------------------------------------------------------
    
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[LOW_RES], 
        -gConfiguration.EffectiveRadius[LOW_RES]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[LOW_RES], 
        gConfiguration.EffectiveRadius[LOW_RES]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[LOW_RES]);
                unsigned int start = dBoundaryCellStart[hash];
                unsigned int end = dBoundaryCellEnd[hash];
                computeBoundaryForceCell(
                    bi,
                    xi,
                    dBoundaryPositions,
                    start,
                    end,
                    LOW_RES
                );
            }
        }
    }

    dAccelerations[3*idx + 0] = fi.x/rhoi + bi.x;
    dAccelerations[3*idx + 1] = fi.y/rhoi - 9.81f + bi.y;
    dAccelerations[3*idx + 2] = fi.z/rhoi + bi.z;

    xc.x /= massSum;
    xc.y /= massSum;
    xc.z /= massSum;

    float3 xd;
    xd.x = xi.x - xc.x;
    xd.y = xi.y - xc.y;
    xd.z = xi.z - xc.z;
    float disticm;

    computeNorm(disticm, xd);

    float ene = 1/(psiSum*psiSum*gConfiguration.EffectiveRadius[resID])*
        (velW.x*velW.x + velW.y*velW.y + velW.z*velW.z);
    
    float maxEne = 300.0f;
    ene = min(ene, maxEne);
    
    if (disticm < 0.0025f)
    {
        dColorValues[idx] = 0.3f;
    }
    else
    {
        dColorValues[idx] = ene/maxEne;    
    }

    int state = dStates[idx];

    if (state == 8 && resID == 0)
    {
    //    dStates[idx] = state | 3;   // mark particle for splitting
    }
}
//------------------------------------------------------------------------------
__global__ void integrateD(
    float* dPositions, 
    float* dVelocities, 
    float* dAccelerations,
    int* dStates,
    float* dBlendCoefficients,
    unsigned int* dActiveIDs,               // array of active particle ids
    unsigned int* dNumParticles,            // counter var for active particles
    float* dPositionsCompl, 
    float* dVelocitiesCompl, 
    int* dStatesCompl,
    float* dBlendCoefficientsCompl,
    unsigned int* dActiveIDsCompl,         
    unsigned int* dNumParticlesCompl,    
    const float* dTempPositions,
    const float* dTempVelocities,
    const float* dDensities,
    const int* dTempStates,
    const float* dTempBlendCoefficients,
    float timeStep,
    unsigned int numParticles,
    int resID
)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    //--------------------------------------------------------------------------
    // update posititions and velocities
    //--------------------------------------------------------------------------

    float3 xi;
    xi.x = dTempPositions[3*idx + 0];
    xi.y = dTempPositions[3*idx + 1];
    xi.z = dTempPositions[3*idx + 2];
    float3 vi;
    vi.x = dTempVelocities[3*idx + 0];
    vi.y = dTempVelocities[3*idx + 1];
    vi.z = dTempVelocities[3*idx + 2];

    vi.x += timeStep*dAccelerations[3*idx + 0]; 
    vi.y += timeStep*dAccelerations[3*idx + 1]; 
    vi.z += timeStep*dAccelerations[3*idx + 2]; 


    // addjust the length of the velocity vector to avoid 
    // small instabilities
    float norm;
    computeNorm(norm, vi);
    float vscale = min(
            norm, 
            0.8f*gConfiguration.EffectiveRadius[resID]/timeStep
        );
    
    vi.x *= vscale/norm;
    vi.y *= vscale/norm;
    vi.z *= vscale/norm;

    xi.x += timeStep*vi.x;
    xi.y += timeStep*vi.y;
    xi.z += timeStep*vi.z;

    // store new position and velocity of the particle
    dPositions[3*idx + 0] = xi.x;
    dPositions[3*idx + 1] = xi.y;
    dPositions[3*idx + 2] = xi.z;

    dVelocities[3*idx + 0] = vi.x;
    dVelocities[3*idx + 1] = vi.y;
    dVelocities[3*idx + 2] = vi.z;

    //--------------------------------------------------------------------------
    // update blend coefficients and states
    //--------------------------------------------------------------------------

    const float c[] = {0.0f, -1.0f, 1.0f, 0.0f}; 

    int state = dTempStates[idx];
    float blendCoeff = dTempBlendCoefficients[idx];
    
    blendCoeff += c[state & 0x03]*gConfiguration.BlendIncrement;

    if (blendCoeff <= 0.0f && (state & 0x03) == 1)
    {
        dBlendCoefficients[idx] = blendCoeff;
        dStates[idx] = state;
        return;
    }

    if (blendCoeff >= 1.0f && (state & 0x03) == 2)
    {
        // if the particle has/reaches a blend coeff of 1.0f or above
        // it stays/becomes a default particle (0)
        state = state & 0x04; 
        blendCoeff = 1.0f;
    }

    // low res particles which are marked for splitting are splitted here
    if ((state & 0x03) == 3 && resID == LOW_RES)
    {
        float dens = dDensities[idx];

        insertHighResParticles(
            dPositionsCompl,
            dVelocitiesCompl,
            dBlendCoefficientsCompl,
            dStatesCompl,
            dActiveIDsCompl,
            dNumParticlesCompl,
            xi,
            vi,
            dens
        );

        state = 0x01; // set state to delete (1) and 
    }

    dBlendCoefficients[idx] = blendCoeff;
    dStates[idx] = state;

    unsigned int i = atomicAdd(dNumParticles, 1);
    dActiveIDs[i] = idx;
}
//------------------------------------------------------------------------------

//==============================================================================
//  HOST code starts here 
//==============================================================================

//------------------------------------------------------------------------------
#define BLOCK_DIMENSIONS_X 256
#define EMPTY_CELL_ID 0xFF
//------------------------------------------------------------------------------

//==============================================================================
//  UTILITY functions start here
//==============================================================================

//------------------------------------------------------------------------------
void computeGridDimensions(
    dim3& gridDimensions, 
    const dim3& blockDimensions,
    unsigned int numParticles
)
{
    // compute the dimensions of the cuda grid for tgiven block dimensions,
    // and the number of particles

    if (numParticles % blockDimensions.x == 0)
    {
        gridDimensions.x = numParticles/blockDimensions.x;
    }
    else
    {
        gridDimensions.x = numParticles/blockDimensions.x + 1;    
    }
    gridDimensions.y = 1;
    gridDimensions.z = 1;
}
//------------------------------------------------------------------------------

//==============================================================================
// SPHParticleData's definition
//==============================================================================

//------------------------------------------------------------------------------
Solver::SPHParticleData::SPHParticleData (
    ParticleData* data, 
    unsigned int numGridCells   // # of grid cells in each direction
)
: 
    Data(data),
    BlockDimensions(BLOCK_DIMENSIONS_X, 1, 1), 
    NumGridCells(numGridCells)
{
    // allocate additional memory for storing density, pressure, acceleration,
    // velocities and hash values for the particles and initialize that data.
    // also allocate memory for the neighbor search as described in the nVidia
    // particles white paper

    CUDA::Alloc<float>(&dDensities, data->MaxParticles);
    CUDA::Alloc<float>(&dPressures, data->MaxParticles);
    CUDA::Alloc<float>(&dAccelerations, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dVelocities, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dTempPositions, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dTempVelocities, 3*data->MaxParticles);
    CUDA::Fill<float>(dDensities, data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dPressures, data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dAccelerations, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dVelocities, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dTempPositions, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dTempVelocities, 3*data->MaxParticles, 0.0f);
    CUDA::Alloc<unsigned int>(&dActiveIDs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dHashs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dCellStart, numGridCells);
    CUDA::Alloc<unsigned int>(&dCellEnd, numGridCells);
    CUDA::Fill<unsigned int>(dActiveIDs, data->MaxParticles, 0, 1);
    CUDA::Fill<unsigned int>(dHashs, data->MaxParticles, 0);
    CUDA::Fill<unsigned int>(dCellStart, numGridCells, 0);
    CUDA::Fill<unsigned int>(dCellEnd, numGridCells, 0);
    CUDA::Alloc<int>(&dTempStates, data->MaxParticles);
    CUDA::Alloc<float>(&dBlendCoefficients, data->MaxParticles);
    CUDA::Alloc<float>(&dTempBlendCoefficients, data->MaxParticles);
    CUDA::Fill<int>(dTempStates, data->MaxParticles, 0);
    CUDA::Fill<float>(dBlendCoefficients, data->MaxParticles, 1.0f);
    CUDA::Fill<float>(dTempBlendCoefficients, data->MaxParticles, 1.0f);
    CUDA::Alloc<unsigned int>(&dNumParticles, 1);
    CUDA::Memcpy<unsigned int>(
        dNumParticles, 
        &Data->NumParticles, 
        1, 
        hipMemcpyHostToDevice
    );

    // compute the number of cuda blocks we need based on the current number
    // of particles and the threads per block we use. Also compute the amount
    // of shared memory we need to compute the values for [dCellStart] and
    // [dCellEnd]
    computeGridDimensions(GridDimensions, BlockDimensions, data->MaxParticles);
    SharedMemSize = sizeof(int)*(BlockDimensions.x + 1);
}
//------------------------------------------------------------------------------
Solver::SPHParticleData::~SPHParticleData ()
{
    // free everything

    CUDA::Free<float>(&dDensities);
    CUDA::Free<float>(&dAccelerations);
    CUDA::Free<float>(&dPressures);
    CUDA::Free<float>(&dVelocities);
    CUDA::Free<float>(&dTempVelocities);
    CUDA::Free<float>(&dTempPositions);

    CUDA::Free<unsigned int>(&dHashs);
    CUDA::Free<unsigned int>(&dCellStart);
    CUDA::Free<unsigned int>(&dCellEnd);

    CUDA::Free<int>(&dTempStates);
    CUDA::Free<float>(&dBlendCoefficients);
    CUDA::Free<float>(&dTempBlendCoefficients);

    CUDA::Free<unsigned int>(&dNumParticles);
}
//------------------------------------------------------------------------------
Solver::BoundaryParticleData::BoundaryParticleData (
    ParticleData* data, 
    unsigned int numGridCells
)
:
    Data(data),
    BlockDimensions(BLOCK_DIMENSIONS_X, 1, 1), 
    NumGridCells(numGridCells)
{
    CUDA::Alloc<unsigned int>(&dHashs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dCellStart, numGridCells);
    CUDA::Alloc<unsigned int>(&dCellEnd, numGridCells);
    CUDA::Fill<unsigned int>(dHashs, data->MaxParticles, 0);
    CUDA::Fill<unsigned int>(dCellStart, numGridCells, 0);
    CUDA::Fill<unsigned int>(dCellEnd, numGridCells, 0);    

    computeGridDimensions(GridDimensions, BlockDimensions, data->NumParticles);
    SharedMemSize = sizeof(int)*(BlockDimensions.x + 1);
}
//------------------------------------------------------------------------------
Solver::BoundaryParticleData::~BoundaryParticleData ()
{
    CUDA::Free<unsigned int>(&dHashs);
    CUDA::Free<unsigned int>(&dCellStart);
    CUDA::Free<unsigned int>(&dCellEnd);
}
//------------------------------------------------------------------------------

//==============================================================================
// Solvers's definition
//==============================================================================

//------------------------------------------------------------------------------
Solver::Solver(
    ParticleData* fluidData, 
    ParticleData* fluidDataHigh,
    ParticleData* boundaryData,
    const SolverConfiguration* configuration
)
: 
    mConfiguration(*configuration)
{
    // store pointer to fluid particles and boundary particles also store
    // a copy of solver configuration

    mFluidData[0] = new SPHParticleData(
            fluidData, 
            Grid::ComputeNumGridCells(configuration->Grid[0])
        );
    mFluidData[1] = new SPHParticleData(
            fluidDataHigh, 
            Grid::ComputeNumGridCells(configuration->Grid[1])
        );
    mBoundaryData = new BoundaryParticleData(
            boundaryData, 
            Grid::ComputeNumGridCells(configuration->Grid[0])
        );

    // compute neighborhood of boundary particles beforehand 
    unsigned int* dBoundaryIDs;
    CUDA::Alloc<unsigned int>(&dBoundaryIDs, mBoundaryData->Data->MaxParticles);
    CUDA::Fill<unsigned int>(dBoundaryIDs, mBoundaryData->Data->MaxParticles, 
        0, 1);

    float* dBoundaryPositions;
    CUDA::Alloc<float>(
        &dBoundaryPositions, 
        3*mBoundaryData->Data->MaxParticles
    );
    
    this->Bind();   // bind first
    mBoundaryData->Data->Map();
    computeHashs<<<mBoundaryData->GridDimensions, 
        mBoundaryData->BlockDimensions>>>(
        mBoundaryData->dHashs, 
        dBoundaryIDs, 
        mBoundaryData->Data->dPositions,
        mBoundaryData->Data->NumParticles,
        0
    );
    thrust::sort_by_key(
        thrust::device_ptr<unsigned int>(mBoundaryData->dHashs),
        thrust::device_ptr<unsigned int>(mBoundaryData->dHashs + 
            mBoundaryData->Data->NumParticles),
        thrust::device_ptr<unsigned int>(dBoundaryIDs)
    );
    CUDA::Memset<unsigned int>(
        mBoundaryData->dCellStart, 
        EMPTY_CELL_ID, 
        mBoundaryData->NumGridCells
    );
    CUDA::Memset<unsigned int>(
        mBoundaryData->dCellEnd, 
        EMPTY_CELL_ID, 
        mBoundaryData->NumGridCells
    );
    reorderComputeCellStartEndBoundaryD<<<mBoundaryData->GridDimensions,
        mBoundaryData->BlockDimensions, 
        mBoundaryData->SharedMemSize>>>(
        mBoundaryData->dCellStart,
        mBoundaryData->dCellEnd,
        dBoundaryPositions,
        mBoundaryData->Data->dPositions,
        dBoundaryIDs,
        mBoundaryData->dHashs,
        mBoundaryData->Data->NumParticles 
    );
    CUDA::Memcpy<float>(
        mBoundaryData->Data->dPositions, 
        dBoundaryPositions, 
        3*mBoundaryData->Data->MaxParticles,
        hipMemcpyDeviceToDevice
    );
    mBoundaryData->Data->Unmap();

    CUDA::Free<unsigned int>(&dBoundaryIDs);
    CUDA::Free<float>(&dBoundaryPositions);

    // set particle states to either low or high
    mFluidData[0]->Data->Map();
    CUDA::Fill<int>(
        mFluidData[0]->Data->dStates, 
        mFluidData[0]->Data->MaxParticles,
        0
    );
    mFluidData[0]->Data->Unmap();
    mFluidData[1]->Data->Map();
    CUDA::Fill<int>(
        mFluidData[1]->Data->dStates, 
        mFluidData[1]->Data->MaxParticles,
        4
    );
    mFluidData[1]->Data->Unmap();
}
//------------------------------------------------------------------------------
Solver::~Solver()
{
    delete mFluidData[LOW_RES];
    delete mFluidData[HIGH_RES];
    delete mBoundaryData;
}
//------------------------------------------------------------------------------
void Solver::Bind() const
{
    // set the configuration of this solver on the device
    CUDA::SafeCall(
        hipMemcpyToSymbol(HIP_SYMBOL(
            gConfiguration), 
            &mConfiguration, 
            sizeof(mConfiguration)
        ), 
        __FILE__, 
        __LINE__ 
    );
}
//------------------------------------------------------------------------------
void Solver::Advance(float timeStep)
{
    //std::cout << mFluidData[LOW_RES]->Data->NumParticles << " " 
    //    << mFluidData[HIGH_RES]->Data->NumParticles << std::endl;

    CUDA::Timer t;
    t.Start();
    mFluidData[LOW_RES]->Data->Map();
    mFluidData[HIGH_RES]->Data->Map();
    mBoundaryData->Data->Map();
    this->computeNeighborhoods(HIGH_RES);
    this->computeNeighborhoods(LOW_RES);
    this->computeDensities(HIGH_RES);
    this->computeDensities(LOW_RES);
    this->computeAccelerations(HIGH_RES);
    this->computeAccelerations(LOW_RES);
    //CUDA::DumpArray<float>(mFluidData[LOW_RES]->Data->dColorValues, mFluidData[LOW_RES]->Data->NumParticles);
    CUDA::Memset<unsigned int>(mFluidData[HIGH_RES]->dNumParticles, 0, 1);
    CUDA::Memset<unsigned int>(mFluidData[LOW_RES]->dNumParticles, 0, 1);
    this->integrate(HIGH_RES, timeStep);
    this->integrate(LOW_RES, timeStep);
    mBoundaryData->Data->Unmap();
    mFluidData[HIGH_RES]->Data->Unmap();
    mFluidData[LOW_RES]->Data->Unmap();
    t.Stop();
    t.DumpElapsed();
    //std::system("pause");
}
//------------------------------------------------------------------------------
void Solver::computeNeighborhoods(int resID)
{
    // reset, then compute cell start end list
    // NOTE: it is important to reset the cell lists, even though zero particles
    // are active, as otherwise the lists would still contain the particle ids
    // of the last particles before complete deletion of the system
    CUDA::Memset<unsigned int>(
        mFluidData[resID]->dCellStart, 
        EMPTY_CELL_ID, 
        mFluidData[resID]->NumGridCells
    );
    CUDA::Memset<unsigned int>(
        mFluidData[resID]->dCellEnd, 
        EMPTY_CELL_ID, 
        mFluidData[resID]->NumGridCells
    );

    CUDA::Memcpy<unsigned int>(
        &mFluidData[resID]->Data->NumParticles,
        mFluidData[resID]->dNumParticles,
        1,
        hipMemcpyDeviceToHost
    );

    if (!mFluidData[resID]->Data->NumParticles)
    {
        return;
    }

    // compute hashs of all particles
    computeHashs<<<mFluidData[resID]->GridDimensions,
        mFluidData[resID]->BlockDimensions>>>(
        mFluidData[resID]->dHashs,
        mFluidData[resID]->dActiveIDs,
        mFluidData[resID]->Data->dPositions,
        mFluidData[resID]->Data->NumParticles,
        resID
    );
        
    // sort the active particle ids by their hash
    thrust::sort_by_key(
        thrust::device_ptr<unsigned int>(mFluidData[resID]->dHashs),
        thrust::device_ptr<unsigned int>(mFluidData[resID]->dHashs + 
            mFluidData[resID]->Data->NumParticles),
        thrust::device_ptr<unsigned int>(mFluidData[resID]->dActiveIDs)
    );

    reorderAndComputeCellStartEndD<<<mFluidData[resID]->GridDimensions, 
        mFluidData[resID]->BlockDimensions, mFluidData[resID]->SharedMemSize>>>(
        mFluidData[resID]->dCellStart,
        mFluidData[resID]->dCellEnd,
        mFluidData[resID]->dTempPositions,
        mFluidData[resID]->dTempVelocities,
        mFluidData[resID]->dTempStates,
        mFluidData[resID]->dTempBlendCoefficients,
        mFluidData[resID]->dActiveIDs,
        mFluidData[resID]->Data->dPositions,
        mFluidData[resID]->dVelocities,
        mFluidData[resID]->Data->dStates,
        mFluidData[resID]->dBlendCoefficients,
        mFluidData[resID]->dHashs,
        mFluidData[resID]->Data->NumParticles
    );

}
//------------------------------------------------------------------------------
void Solver::computeDensities (int resID)
{
    if (!mFluidData[resID]->Data->NumParticles)
    {
        return;
    }

    computeDensitiesPressuresD<<<mFluidData[resID]->GridDimensions, 
        mFluidData[resID]->BlockDimensions>>>(
        mFluidData[resID]->dDensities,
        mFluidData[resID]->dPressures,
        mFluidData[resID]->dTempPositions,
        mFluidData[resID]->dTempBlendCoefficients,
        mFluidData[resID]->dCellStart,
        mFluidData[resID]->dCellEnd,
        mFluidData[COMPL(resID)]->dTempPositions,
        mFluidData[COMPL(resID)]->dTempBlendCoefficients,
        mFluidData[COMPL(resID)]->dCellStart,
        mFluidData[COMPL(resID)]->dCellEnd,
        mFluidData[resID]->Data->NumParticles,
        resID
    );
}
//------------------------------------------------------------------------------
void Solver::computeAccelerations(int resID)
{
    if (!mFluidData[resID]->Data->NumParticles)
    {
        return;
    }

    computeAccelerationsD<<<mFluidData[resID]->GridDimensions, 
        mFluidData[resID]->BlockDimensions>>>(
        mFluidData[resID]->dAccelerations,
        mFluidData[resID]->Data->dColorValues,
        mFluidData[resID]->dDensities,
        mFluidData[resID]->dPressures,
        mFluidData[resID]->dTempPositions,
        mFluidData[resID]->dTempVelocities,
        mFluidData[resID]->dTempBlendCoefficients,
        mFluidData[resID]->dTempStates,
        mFluidData[resID]->dCellStart,
        mFluidData[resID]->dCellEnd,
        mFluidData[COMPL(resID)]->dDensities,
        mFluidData[COMPL(resID)]->dPressures,
        mFluidData[COMPL(resID)]->dTempPositions,
        mFluidData[COMPL(resID)]->dTempVelocities,
        mFluidData[COMPL(resID)]->dTempBlendCoefficients,
        mFluidData[COMPL(resID)]->dCellStart,
        mFluidData[COMPL(resID)]->dCellEnd,
        mBoundaryData->Data->dPositions,
        mBoundaryData->dCellStart,
        mBoundaryData->dCellEnd,
        mFluidData[resID]->Data->NumParticles,
        resID
    );
}
//------------------------------------------------------------------------------
void Solver::integrate(int resID, float timeStep)
{
    if (!mFluidData[resID]->Data->NumParticles)
    {
        return;
    }

    integrateD<<<mFluidData[resID]->GridDimensions, 
        mFluidData[resID]->BlockDimensions>>>(
        mFluidData[resID]->Data->dPositions,
        mFluidData[resID]->dVelocities,
        mFluidData[resID]->dAccelerations,
        mFluidData[resID]->Data->dStates,
        mFluidData[resID]->dBlendCoefficients,
        mFluidData[resID]->dActiveIDs,
        mFluidData[resID]->dNumParticles,
        mFluidData[COMPL(resID)]->Data->dPositions,
        mFluidData[COMPL(resID)]->dVelocities,
        mFluidData[COMPL(resID)]->Data->dStates,
        mFluidData[COMPL(resID)]->dBlendCoefficients,
        mFluidData[COMPL(resID)]->dActiveIDs,
        mFluidData[COMPL(resID)]->dNumParticles,
        mFluidData[resID]->dTempPositions,
        mFluidData[resID]->dTempVelocities,
        mFluidData[resID]->dDensities,
        mFluidData[resID]->dTempStates,
        mFluidData[resID]->dTempBlendCoefficients,
        timeStep,
        mFluidData[resID]->Data->NumParticles,
        resID
    );
}
//------------------------------------------------------------------------------

#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
//  Solver.cu
//------------------------------------------------------------------------------
#include "Solver.h"
//------------------------------------------------------------------------------

//==============================================================================
//  CUDA DEVICE code starts here 
//==============================================================================

//------------------------------------------------------------------------------
__constant__ SolverConfiguration gConfiguration;    // current solver's config.
//------------------------------------------------------------------------------
enum
{
    // particle state
    PS_DEFAULT = 0,
    PS_DELETE,          // particle is marked for deletion
    PS_INSERT           // particle is marked for insertion
};
//------------------------------------------------------------------------------
#define BLOCK_DIMENSIONS_X 256
#define EMPTY_CELL_ID 0xFFFFFFFF
//------------------------------------------------------------------------------

//==============================================================================
// UTLITY device kernels definition
//==============================================================================

//------------------------------------------------------------------------------
__device__ void computeCoordinatesOff (
    int3& coordinate,            // out: coordinate for [position]
    float3 position,
    const Grid& grid,
    float offset
)
{
    // compute the coordinates of a point in space with respect to the given 
    // grid

    coordinate.x = (int)((position.x + offset - grid.Origin.x)/grid.Spacing);
    coordinate.y = (int)((position.y + offset - grid.Origin.y)/grid.Spacing);
    coordinate.z = (int)((position.z + offset - grid.Origin.z)/grid.Spacing);

    // clamp coordinates if neccessary
    coordinate.x = max(0, min(coordinate.x, grid.Dimensions.x - 1));
    coordinate.y = max(0, min(coordinate.y, grid.Dimensions.y - 1));
    coordinate.z = max(0, min(coordinate.z, grid.Dimensions.z - 1));
}
//------------------------------------------------------------------------------
__device__ void computeCoordinates (
    int3& coordinate,            // out: coordinate for [position]
    float3 position,
    const Grid& grid
)
{
    // compute the coordinates of a point in space with respect to the given 
    // grid

    coordinate.x = (int)((position.x - grid.Origin.x)/grid.Spacing);
    coordinate.y = (int)((position.y - grid.Origin.y)/grid.Spacing);
    coordinate.z = (int)((position.z - grid.Origin.z)/grid.Spacing);

    // clamp coordinates if neccessary
    coordinate.x = max(0, min(coordinate.x, grid.Dimensions.x - 1));
    coordinate.y = max(0, min(coordinate.y, grid.Dimensions.y - 1));
    coordinate.z = max(0, min(coordinate.z, grid.Dimensions.z - 1));
}
//------------------------------------------------------------------------------
__device__ void computeHash (
    unsigned int& hash, 
    const int3& coordinate,
    const Grid& grid
)
{
    // compute the hash for a grid given a coordinate within the grid

    hash = coordinate.x + grid.Dimensions.x*
        (coordinate.y + grid.Dimensions.y*coordinate.z);
}
//------------------------------------------------------------------------------
__device__ void computeHash (
    unsigned int& hash, 
    float3 position,
    const Grid& grid
)
{
    // compute the hash for a grid given a position in world space, by first
    // conputing the coordinate in [grid], and then computing the hash.

    int3 coordinate;
    computeCoordinates(coordinate, position, grid);
    computeHash(hash, coordinate, grid);
}
//------------------------------------------------------------------------------
__device__ inline void computeNorm (float& norm, const float3& a)
{
    norm = sqrt(a.x*a.x + a.y*a.y + a.z*a.z);
}
//------------------------------------------------------------------------------
__device__ inline void computeDistance (
    float& dist, 
    const float3& a, 
    const float3& b
)
{
    float3 d;
    d.x = a.x - b.x;
    d.y = a.y - b.y;
    d.z = a.z - b.z;
    computeNorm(dist, d); 
}
//------------------------------------------------------------------------------
__device__ inline void evaluatePoly6Kernel (
    float& res,  // [out] result of evaluation
    float d,     // distance between two particles
    float h      // effective radius 
)
{
    // evaluate Muellers Poly6 Kernel

    float hhh = h*h*h;
    float coeff = 315.0f/(64.0f*M_PI*hhh*hhh*hhh);

    if (d < h)
    {
        float a = h*h - d*d;
        res = coeff*a*a*a;
    }
    else
    {
        res = 0.0f;    
    }
}
//------------------------------------------------------------------------------
__device__ inline void evaluateSpikyKernelGradient (
    float3& grad,
    const float3& xij,
    float h
)
{
    float norm = 0.0f;
    computeNorm(norm, xij);
    
    if ((norm == 0.0f) || (norm > h))
    {
        grad.x = 0.0f;
        grad.y = 0.0f;
        grad.z = 0.0f;
        return;
    } 

    float hhh = h*h*h;
    float a = -45.0f/(M_PI*hhh*hhh)*(h - norm)*(h - norm);

    grad.x = a*xij.x/norm;
    grad.y = a*xij.y/norm;
    grad.z = a*xij.z/norm;
}
//------------------------------------------------------------------------------
__device__ inline void evaluateViscosityKernelLaplacian (
    float& lapl,
    float dist,
    float h
)
{
    if (dist < h)
    {
        float hhh = h*h*h;
        float coeff = 45.0f/(M_PI*hhh*hhh);
        lapl = coeff*(h - dist);
        return;
    }
    else
    {
        return;
    }
}
//------------------------------------------------------------------------------
__device__ inline void evaluateBoundaryWeight (
    float& weight, 
    float dist, 
    float h
)
{
    float q = 2.0f*dist/h;
    float coeff = 0.02f*gConfiguration.SpeedSound*gConfiguration.SpeedSound/
        dist;

    if (q < 2.0f/3.0f)
    {
        weight = coeff*2.0f/3.0f;
    } 
    else if (q < 1.0f)
    {
        weight = coeff*(2.0f*q - 3.0f/2.0f*q*q);
    }
    else if (q < 2.0f)
    {
        float a = 2.0f - q;
        weight = coeff*0.5f*a*a;
    }
    else
    {
        weight = 0.0f;
    }
}
//------------------------------------------------------------------------------
__device__ inline void computeDensityCell (
    float& rhoi,                 // [out] density of particle i 
    const float3& xi,            // position of particle i
    const float* dPositions,     
    unsigned int start,
    unsigned int end,
    float effectiveRadius
)
{
    // add up density contribution form particle in this cell ([start], [end])
    // to the density of the particle i [rhoi]. (in fact only the kernel 
    // weights are added up, mass is multiplied in the callee, to safe
    // operations)
    
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float dist;
        computeDistance(dist, xi, xj);

        if (dist < effectiveRadius)
        {
            float weight = 0.0f;
            evaluatePoly6Kernel(
                weight, 
                dist, 
                effectiveRadius
            );
            rhoi += weight;
        }
            
    }
  
}
//------------------------------------------------------------------------------
__device__ inline void computeDensityCellComplement (
    float& rhoi,                 // [out] density of particle i 
    const float3& xi,            // position of particle i
    const float* dPositions,     // high resolution positions
    unsigned int start,
    unsigned int end
)
{
    // add up density contribution form particle in this cell ([start], [end])
    // to the density of the particle i [rhoi]. (in fact only the kernel 
    // weights are added up, mass is multiplied in the callee, to safe
    // operations)
    
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float dist;
        computeDistance(dist, xi, xj);

        if (dist < gConfiguration.EffectiveRadius[0])
        {
            float weight0 = 0.0f;
            float weight1 = 0.0f;

            evaluatePoly6Kernel(
                weight0, 
                dist, 
                gConfiguration.EffectiveRadius[0]
            );
            evaluatePoly6Kernel(
                weight1, 
                dist, 
                gConfiguration.EffectiveRadius[1]
            );
            rhoi += 0.5f*(weight0 + weight1);
        }
            
    }
  
}
//------------------------------------------------------------------------------
__device__ inline void computeAccelerationCell (
    float3& fi,
    float rhoi,
    float pi,
    const float3& xi,        
    const float3& vi,    
    const float* dDensities,
    const float* dPressures,
    const float* dPositions,     
    const float* dVelocities,
    unsigned int start,
    unsigned int end,
    unsigned char res           // res of the particle 0 == low, 1 == high
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float3 vj;
        vj.x = dVelocities[3*j + 0];
        vj.y = dVelocities[3*j + 1];
        vj.z = dVelocities[3*j + 2];
        float rhoj = dDensities[j];
        float pj = dPressures[j];
        float dist;
        float3 xij;
        xij.x = xi.x - xj.x; 
        xij.y = xi.y - xj.y; 
        xij.z = xi.z - xj.z; 
        computeNorm(dist, xij);
        
        if (dist != 0.0f && dist < gConfiguration.EffectiveRadius[res])
        {
            // evaluate the pressure force partice j exerts on particle i
            float coeffP = -rhoi*gConfiguration.FluidParticleMass[res]*
                (pi/(rhoi*rhoi) + pj/(rhoj*rhoj));
            float3 grad;
            evaluateSpikyKernelGradient(
                grad, 
                xij,
                gConfiguration.EffectiveRadius[res]
            );
            fi.x += coeffP*grad.x;
            fi.y += coeffP*grad.y;
            fi.z += coeffP*grad.z;

            // evaluate the viscosity force partice j exerts on particle i
            float coeffV = gConfiguration.Viscosity*
                gConfiguration.FluidParticleMass[res]/rhoj;
            float lapl = 0.0f;
            evaluateViscosityKernelLaplacian(
                lapl, 
                dist, 
                gConfiguration.EffectiveRadius[res]
            );
            float3 vji;
            vji.x = vj.x - vi.x;
            vji.y = vj.y - vi.y;
            vji.z = vj.z - vi.z;
            fi.x += coeffV*vji.x*lapl;
            fi.y += coeffV*vji.y*lapl;
            fi.z += coeffV*vji.z*lapl;

            // evaluate the surface tension force partice j exerts on particle i
            float weight;
            evaluatePoly6Kernel(
                weight, 
                dist, 
                gConfiguration.EffectiveRadius[res]
            );
            float coeffT = -weight*gConfiguration.FluidParticleMass[res]*
                gConfiguration.TensionCoefficient;
        
            fi.x += coeffT*xij.x;
            fi.y += coeffT*xij.y;
            fi.z += coeffT*xij.z;
        }

    }

}
//------------------------------------------------------------------------------
__device__ inline void computeAccelerationCellComplement (
    float3& fi,
    float rhoi,
    float pi,
    const float3& xi,        
    const float3& vi,    
    const float* dDensities,
    const float* dPressures,
    const float* dPositions,     
    const float* dVelocities,
    unsigned int start,
    unsigned int end,
    unsigned char res           // res of the particle 0 == low, 1 == high
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float3 vj;
        vj.x = dVelocities[3*j + 0];
        vj.y = dVelocities[3*j + 1];
        vj.z = dVelocities[3*j + 2];
        float rhoj = dDensities[j];
        float pj = dPressures[j];
        float dist;
        float3 xij;
        xij.x = xi.x - xj.x; 
        xij.y = xi.y - xj.y; 
        xij.z = xi.z - xj.z; 
        computeNorm(dist, xij);
        
        if (dist != 0.0f && dist < gConfiguration.EffectiveRadius[0])
        {
            // evaluate the pressure force partice j exerts on particle i
            float coeffP = -rhoi*gConfiguration.FluidParticleMass[res]*
                (pi/(rhoi*rhoi) + pj/(rhoj*rhoj));
            float3 grad0;
            float3 grad1;
            evaluateSpikyKernelGradient(
                grad0, 
                xij,
                gConfiguration.EffectiveRadius[res]
            );
            evaluateSpikyKernelGradient(
                grad1, 
                xij,
                gConfiguration.EffectiveRadius[res]
            );
            fi.x += coeffP*0.5f*(grad0.x + grad1.x);
            fi.y += coeffP*0.5f*(grad0.y + grad1.y);
            fi.z += coeffP*0.5f*(grad0.z + grad1.z);

            // evaluate the viscosity force partice j exerts on particle i
            float coeffV = gConfiguration.Viscosity*
                gConfiguration.FluidParticleMass[res]/rhoj;
            float lapl0 = 0.0f;
            float lapl1 = 0.0f;
            evaluateViscosityKernelLaplacian(
                lapl0, 
                dist, 
                gConfiguration.EffectiveRadius[res]
            );
            evaluateViscosityKernelLaplacian(
                lapl1, 
                dist, 
                gConfiguration.EffectiveRadius[res]
            );
            float3 vji;
            vji.x = vj.x - vi.x;
            vji.y = vj.y - vi.y;
            vji.z = vj.z - vi.z;
            fi.x += coeffV*vji.x*0.5f*(lapl0 + lapl1);
            fi.y += coeffV*vji.y*0.5f*(lapl0 + lapl1);
            fi.z += coeffV*vji.z*0.5f*(lapl0 + lapl1);

            // evaluate the surface tension force partice j exerts on particle i
            float weight0 = 0.0f;
            float weight1 = 0.0f;
            evaluatePoly6Kernel(
                weight0, 
                dist, 
                gConfiguration.EffectiveRadius[0]
            );
            evaluatePoly6Kernel(
                weight1, 
                dist, 
                gConfiguration.EffectiveRadius[1]
            );
            float coeffT = -0.5f*(weight0 + weight1)*
                gConfiguration.FluidParticleMass[res]*
                gConfiguration.TensionCoefficient;
        
            fi.x += coeffT*xij.x;
            fi.y += coeffT*xij.y;
            fi.z += coeffT*xij.z;
        }

    }

}
//------------------------------------------------------------------------------
__device__ void computeBoundaryForceCell (
    float3& bi,
    const float3& xi,
    const float* dPositions,     
    unsigned int start,
    unsigned int end,
    unsigned char res
)
{
    for (unsigned int j = start; j < end; j++)
    {
        float3 xj;
        xj.x = dPositions[3*j + 0];
        xj.y = dPositions[3*j + 1];
        xj.z = dPositions[3*j + 2];
        float3 xij;
        xij.x = xi.x - xj.x;
        xij.y = xi.y - xj.y;
        xij.z = xi.z - xj.z;
        float dist;
        computeNorm(dist, xij); 

        if (dist < gConfiguration.EffectiveRadius[res])
        {
            float weight = 0.0f;
            evaluateBoundaryWeight(
                weight, 
                dist, 
                gConfiguration.EffectiveRadius[res]
            );
            weight*= gConfiguration.BoundaryParticleMass/
                (gConfiguration.FluidParticleMass[res] + 
                gConfiguration.BoundaryParticleMass);
            bi.x += weight*xij.x/dist;
            bi.y += weight*xij.y/dist;
            bi.z += weight*xij.z/dist;
        }
    }
}
//------------------------------------------------------------------------------

//==============================================================================
// GLOBAL device kernel definitions
//==============================================================================

//------------------------------------------------------------------------------
__global__ void computeHashsLowD
(
    unsigned int* dHashs,           // hash values of each particle
    unsigned int* dActiveIDs,       // array of active particle ids
    const float* dPositions,        // positions of each particle 
    unsigned int numParticles       // number of ids in the id array
)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    unsigned int id = dActiveIDs[idx];

    float3 pos;
    pos.x = dPositions[3*id + 0];
    pos.y = dPositions[3*id + 1];
    pos.z = dPositions[3*id + 2];

    computeHash(dHashs[idx], pos, gConfiguration.Grid[0]);
};
//------------------------------------------------------------------------------
__global__ void computeHashsHighD
(
    unsigned int* dHashs,           // hash values of each particle
    unsigned int* dActiveIDs,       // array of active particle ids
    const float* dPositions,        // positions of each particle 
    unsigned int numParticles       // number of ids in the id array
)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    unsigned int id = dActiveIDs[idx];

    float3 pos;
    pos.x = dPositions[3*id + 0];
    pos.y = dPositions[3*id + 1];
    pos.z = dPositions[3*id + 2];

    computeHash(dHashs[idx], pos, gConfiguration.Grid[1]);
};
//------------------------------------------------------------------------------
__global__ void reorderComputeCellStartEndBoundaryD
(
    unsigned int* dCellStart,
    unsigned int* dCellEnd,
    float* dTempPositions,
    const float* dPositions,
    const unsigned int* dSortedIDs,
    const unsigned int* dHashs,
    unsigned int numParticles
)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles) 
    {
        return;
    }

    // reorder
    unsigned int id = dSortedIDs[idx];
    dTempPositions[3*idx + 0] = dPositions[3*id + 0];
    dTempPositions[3*idx + 1] = dPositions[3*id + 1];
    dTempPositions[3*idx + 2] = dPositions[3*id + 2];

    // compute cell start end
    int hash = dHashs[idx];
    sharedHash[threadIdx.x + 1] = hash;
        
    if (idx > 0 && threadIdx.x == 0) 
    {
        sharedHash[0] = dHashs[idx - 1];
    }

    __syncthreads();

    if (idx == 0 || hash != sharedHash[threadIdx.x])
    {
        dCellStart[hash] = idx;
        
        if (idx > 0) 
        {
            dCellEnd[sharedHash[threadIdx.x]] = idx;
        }
    }

    if (idx == numParticles - 1)
    {
        dCellEnd[hash] = idx + 1;
    }
}
//------------------------------------------------------------------------------
__global__ void reorderAndComputeCellStartEndD
(
    unsigned int* dCellStart,
    unsigned int* dCellEnd,
    float* dTempPositions,
    float* dTempVelocities,
    float* dTempBlendCoefficients,
    unsigned char* dTempStates,
    unsigned int* dSortedIDs,
    const float* dPositions,
    const float* dVelocities,
    const float* dBlendVelocities,
    const unsigned char* dStates,
    const unsigned int* dHashs,
    unsigned int numParticles
)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles) 
    {
        return;
    }
    
    // reorder
    unsigned int id = dSortedIDs[idx];
    dTempPositions[3*idx + 0] = dPositions[3*id + 0]; 
    dTempPositions[3*idx + 1] = dPositions[3*id + 1]; 
    dTempPositions[3*idx + 2] = dPositions[3*id + 2]; 
    dTempVelocities[3*idx + 0] = dVelocities[3*id + 0]; 
    dTempVelocities[3*idx + 1] = dVelocities[3*id + 1]; 
    dTempVelocities[3*idx + 2] = dVelocities[3*id + 2];
    dTempBlendCoefficients[idx] = dBlendVelocities[id];
    dTempStates[idx] = dStates[id];
    dSortedIDs[idx] = idx;

    // compute cell start end ids
    int hash = dHashs[idx];
    sharedHash[threadIdx.x + 1] = hash;
        
    if (idx > 0 && threadIdx.x == 0) 
    {
        sharedHash[0] = dHashs[idx - 1];
    }

    __syncthreads();

    if (idx == 0 || hash != sharedHash[threadIdx.x])
    {
        dCellStart[hash] = idx;
        
        if (idx > 0) 
        {
            dCellEnd[sharedHash[threadIdx.x]] = idx;
        }
    }

    if (idx == numParticles - 1)
    {
        dCellEnd[hash] = idx + 1;
    }
}
//------------------------------------------------------------------------------
__global__ void computeDensitiesPressuresLowD (
    float* dDensities,              // [out] computed densities
    float* dPressures,              // [out] computed pressures
    const float* dPositions,
    const float* dPositionsHigh,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const unsigned int* dCellStartHigh,
    const unsigned int* dCellEndHigh,
    unsigned int numParticles
)
{
    // compute densities and pressures for the low resolution particles.

    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];

    float rhoi = 0.0f;
    float rhoiComplement = 0.0f;
    int3 cs, ce;
    int3 cc;

    //--------------------------------------------------------------------------
    // compute the density contribution of the low res domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[0], 
        -gConfiguration.EffectiveRadius[0]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[0], 
        gConfiguration.EffectiveRadius[0]
    );


    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[0]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeDensityCell(
                    rhoi,
                    xi,
                    dPositions,
                    start,
                    end,
                    gConfiguration.EffectiveRadius[0]
                );
            }
        }
    }
    //--------------------------------------------------------------------------

    //--------------------------------------------------------------------------
    // compute the density contribution of the high res domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[1], 
        -gConfiguration.EffectiveRadius[0]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[1], 
        gConfiguration.EffectiveRadius[0]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[1]);
                unsigned int start = dCellStartHigh[hash];
                unsigned int end = dCellStartHigh[hash];

                computeDensityCellComplement(
                    rhoiComplement,
                    xi,
                    dPositionsHigh,
                    start,
                    end
                );
            }
        }
    }
    //--------------------------------------------------------------------------    

    rhoi = rhoi*gConfiguration.FluidParticleMass[0] + 
        rhoiComplement*gConfiguration.FluidParticleMass[1];
    dDensities[idx] = rhoi;
    dPressures[idx] = gConfiguration.BulkModulus*
        (rhoi - gConfiguration.RestDensity);
}
//------------------------------------------------------------------------------
__global__ void computeDensitiesPressuresHighD (
    float* dDensities,              // [out] computed densities
    float* dPressures,              // [out] computed pressures
    const float* dPositions,
    const float* dPositionsLow,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const unsigned int* dCellStartLow,
    const unsigned int* dCellEndLow,
    unsigned int numParticles
)
{
    // compute densities and pressures for the high resolution particles.

    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];

    float rhoi = 0.0f;
    float rhoiComplement = 0.0f;
    int3 cs, ce;
    int3 cc;

    //--------------------------------------------------------------------------
    // compute the density contribution of the high res domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[1], 
        -gConfiguration.EffectiveRadius[1]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[1], 
        gConfiguration.EffectiveRadius[1]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[1]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeDensityCell(
                    rhoi,
                    xi,
                    dPositions,
                    start,
                    end,
                    gConfiguration.EffectiveRadius[1]
                );

                rhoi += end-EMPTY_CELL_ID;
            }
        }
    }
    //--------------------------------------------------------------------------

    //--------------------------------------------------------------------------
    // compute the density contribution of the low res domain
    //computeCoordinatesOff(
    //    cs, 
    //    xi, 
    //    gConfiguration.Grid[0], 
    //    -gConfiguration.EffectiveRadius[0]
    //);
    //computeCoordinatesOff(
    //    ce, 
    //    xi, 
    //    gConfiguration.Grid[0], 
    //    gConfiguration.EffectiveRadius[0]
    //);

    //for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    //{
    //    for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
    //    {
    //        for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
    //        {
    //            unsigned int hash;
    //            computeHash(hash, cc, gConfiguration.Grid[0]);
    //            unsigned int start = dCellStartLow[hash];
    //            unsigned int end = dCellStartLow[hash];

    //            computeDensityCellComplement(
    //                rhoiComplement,
    //                xi,
    //                dPositionsLow,
    //                start,
    //                end
    //            );
    //        }
    //    }
    //}
    //--------------------------------------------------------------------------    

    rhoi = rhoi*gConfiguration.FluidParticleMass[1] + 
        rhoiComplement*gConfiguration.FluidParticleMass[0];
    dDensities[idx] = rhoi;
    dPressures[idx] = gConfiguration.BulkModulus*
        (rhoi - gConfiguration.RestDensity);
}
//------------------------------------------------------------------------------
__global__ void computeAccelerationsAndUpdateStatesLowD (
    float* dAccelerations,
    unsigned char* dStates,
    const float* dDensities,              
    const float* dPressures,
    const float* dPositions,
    const float* dVelocities,
    const float* dDensitiesHigh,              
    const float* dPressuresHigh,
    const float* dPositionsHigh,
    const float* dVelocitiesHigh,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const unsigned int* dCellStartHigh,
    const unsigned int* dCellEndHigh,
    const float* dBoundaryPositions,
    const unsigned int* dBoundaryCellStart,
    const unsigned int* dBoundaryCellEnd,
    unsigned int numParticles
)
{
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];
    float3 vi;
    vi.x = dVelocities[3*idx + 0];
    vi.y = dVelocities[3*idx + 1];
    vi.z = dVelocities[3*idx + 2];
    float rhoi = dDensities[idx];
    float pi = dPressures[idx];
    float3 fi;
    fi.x = 0.0;
    fi.y = 0.0;
    fi.z = 0.0;
    float3 bi;
    bi.x = 0.0f;
    bi.y = 0.0f;
    bi.z = 0.0f;
    int3 cs, ce;
    int3 cc;

    // compute force contribution from the same domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[0], 
        -gConfiguration.EffectiveRadius[0]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[0], 
        gConfiguration.EffectiveRadius[0]
    );
    
    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[0]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeAccelerationCell(
                    fi,
                    rhoi,
                    pi,
                    xi,
                    vi,
                    dDensities,
                    dPressures,
                    dPositions,
                    dVelocities,
                    start,
                    end,
                    0
                );
            }
        }
    }

    // compute force contribution from the complement domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[1], 
        -gConfiguration.EffectiveRadius[1]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[1], 
        gConfiguration.EffectiveRadius[1]
    );
    
    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[1]);
                unsigned int start = dCellStartHigh[hash];
                unsigned int end = dCellEndHigh[hash];

                computeAccelerationCellComplement(
                    fi,
                    rhoi,
                    pi,
                    xi,
                    vi,
                    dDensitiesHigh,
                    dPressuresHigh,
                    dPositionsHigh,
                    dVelocitiesHigh,
                    start,
                    end,
                    1
                );
            }
        }
    }

    // compute force contribution from the boundary
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[0], 
        -gConfiguration.EffectiveRadius[0]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[0], 
        gConfiguration.EffectiveRadius[0]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[0]);
                unsigned int start = dBoundaryCellStart[hash];
                unsigned int end = dBoundaryCellEnd[hash];
                computeBoundaryForceCell(
                    bi,
                    xi,
                    dBoundaryPositions,
                    start,
                    end,
                    0
                );
            }
        }
    }

    dAccelerations[3*idx + 0] = fi.x/rhoi + bi.x;
    dAccelerations[3*idx + 1] = fi.y/rhoi - 9.81f + bi.y;
    dAccelerations[3*idx + 2] = fi.z/rhoi + bi.z;

    // update states
    //if (xi.x > 0.5f)
    //{
    //    dStates[idx] = PS_DELETE;
    //}
}
//------------------------------------------------------------------------------
__global__ void computeAccelerationsAndUpdateStatesHighD (
    float* dAccelerations,
    unsigned char* dStates,
    const float* dDensities,              
    const float* dPressures,
    const float* dPositions,
    const float* dVelocities,
    const float* dDensitiesLow,              
    const float* dPressuresLow,
    const float* dPositionsLow,
    const float* dVelocitiesLow,
    const unsigned int* dCellStart,
    const unsigned int* dCellEnd,
    const unsigned int* dCellStartLow,
    const unsigned int* dCellEndLow,
    const float* dBoundaryPositions,
    const unsigned int* dBoundaryCellStart,
    const unsigned int* dBoundaryCellEnd,
    unsigned int numParticles
)
{
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dPositions[3*idx + 0];
    xi.y = dPositions[3*idx + 1];
    xi.z = dPositions[3*idx + 2];
    float3 vi;
    vi.x = dVelocities[3*idx + 0];
    vi.y = dVelocities[3*idx + 1];
    vi.z = dVelocities[3*idx + 2];
    float rhoi = dDensities[idx];
    float pi = dPressures[idx];
    float3 fi;
    fi.x = 0.0;
    fi.y = 0.0;
    fi.z = 0.0;
    float3 bi;
    bi.x = 0.0f;
    bi.y = 0.0f;
    bi.z = 0.0f;
    int3 cs, ce;
    int3 cc;

    // compute force contribution from the same domain
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[1], 
        -gConfiguration.EffectiveRadius[1]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[1], 
        gConfiguration.EffectiveRadius[1]
    );
    
    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[1]);
                unsigned int start = dCellStart[hash];
                unsigned int end = dCellEnd[hash];

                computeAccelerationCell(
                    fi,
                    rhoi,
                    pi,
                    xi,
                    vi,
                    dDensities,
                    dPressures,
                    dPositions,
                    dVelocities,
                    start,
                    end,
                    1
                );
            }
        }
    }

    // compute force contribution from the complement domain
    //computeCoordinatesOff(
    //    cs, 
    //    xi, 
    //    gConfiguration.Grid[0], 
    //    -gConfiguration.EffectiveRadius[0]
    //);
    //computeCoordinatesOff(
    //    ce, 
    //    xi, 
    //    gConfiguration.Grid[0], 
    //    gConfiguration.EffectiveRadius[0]
    //);
    //
    //for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    //{
    //    for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
    //    {
    //        for (cc.x = cs.x; cc.x <= ce.x; cc.x++)
    //        {
    //            unsigned int hash;
    //            computeHash(hash, cc, gConfiguration.Grid[0]);
    //            unsigned int start = dCellStartLow[hash];
    //            unsigned int end = dCellEndLow[hash];

    //            computeAccelerationCellComplement(
    //                fi,
    //                rhoi,
    //                pi,
    //                xi,
    //                vi,
    //                dDensitiesLow,
    //                dPressuresLow,
    //                dPositionsLow,
    //                dVelocitiesLow,
    //                start,
    //                end,
    //                0
    //            );
    //        }
    //    }
    //}

    // compute force contribution from the boundary
    computeCoordinatesOff(
        cs, 
        xi, 
        gConfiguration.Grid[0], 
        -gConfiguration.EffectiveRadius[0]
    );
    computeCoordinatesOff(
        ce, 
        xi, 
        gConfiguration.Grid[0], 
        gConfiguration.EffectiveRadius[0]
    );

    for (cc.z = cs.z; cc.z <= ce.z; cc.z++)
    {
        for (cc.y = cs.y; cc.y <= ce.y; cc.y++)
        {
            for (cc.x  = cs.x; cc.x <= ce.x; cc.x++)
            {
                unsigned int hash;
                computeHash(hash, cc, gConfiguration.Grid[0]);
                unsigned int start = dBoundaryCellStart[hash];
                unsigned int end = dBoundaryCellEnd[hash];
                computeBoundaryForceCell(
                    bi,
                    xi,
                    dBoundaryPositions,
                    start,
                    end,
                    1
                );
            }
        }
    }

    dAccelerations[3*idx + 0] = fi.x/rhoi + bi.x;
    dAccelerations[3*idx + 1] = fi.y/rhoi - 9.81f + bi.y;
    dAccelerations[3*idx + 2] = fi.z/rhoi + bi.z;
}
//------------------------------------------------------------------------------
__global__ void updateSystemD (
    float* dPositions, 
    float* dVelocities, 
    float* dBlendCoefficients,
    unsigned char* dStates,
    unsigned int* dActiveIDs,
    unsigned int* dNumParticles,
    const float* dTempPositions,
    const float* dTempVelocities,
    const float* dTempBlendCoefficients,
    const unsigned char* dTempStates,
    const float* dAccelerations,
    float timeStep,
    unsigned int numParticles
)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    float3 xi;
    xi.x = dTempPositions[3*idx + 0];
    xi.y = dTempPositions[3*idx + 1];
    xi.z = dTempPositions[3*idx + 2];
    float3 vi;
    vi.x = dTempVelocities[3*idx + 0];
    vi.y = dTempVelocities[3*idx + 1];
    vi.z = dTempVelocities[3*idx + 2];

    // update position and velocity of the particle
    vi.x += timeStep*dAccelerations[3*idx + 0]; 
    vi.y += timeStep*dAccelerations[3*idx + 1]; 
    vi.z += timeStep*dAccelerations[3*idx + 2]; 
    xi.x += timeStep*vi.x;
    xi.y += timeStep*vi.y;
    xi.z += timeStep*vi.z;

    // store new position and velocity of the particle
    dPositions[3*idx + 0] = xi.x;
    dPositions[3*idx + 1] = xi.y;
    dPositions[3*idx + 2] = xi.z;

    dVelocities[3*idx + 0] = vi.x;
    dVelocities[3*idx + 1] = vi.y;
    dVelocities[3*idx + 2] = vi.z;

    // update blend values

    // [c] is a mapping functions, since for default particles blendvalues
    // should not be changed (i.e. 0 => 0.0f), for particles for deletion
    // the blend values should be substracted by the blend increment 
    // (i.e. 1 => -1.0), for particles marked for insertion blend values
    // should be added by the blend increment (i.e. 2 => 1.0f)
    const float c[] = {0.0f, -1.0f, 1.0f};

    float blendVal = dTempBlendCoefficients[idx];
    unsigned char state = dTempStates[idx];

    blendVal += c[state]*gConfiguration.BlendIncrement;

    dBlendCoefficients[idx] = blendVal;
    dStates[idx] = state;

    // if the blend value of the particle is greater than zero (i.e. it 
    // contributes to the system) add it to the particle list for computations
    // in the next time step. Otherwise, disregard the particle.
    if (blendVal > 0.0f)
    {
        // (atomic) add one to the particle count
        unsigned int old = atomicAdd(dNumParticles, 1);
        dActiveIDs[old] = idx;
    }
}
//------------------------------------------------------------------------------

//==============================================================================
//  HOST code starts here 
//==============================================================================

//==============================================================================
//  UTILITY functions start here
//==============================================================================

//------------------------------------------------------------------------------
void computeGridDimensions 
(
    dim3& gridDimensions, 
    const dim3& blockDimensions,
    unsigned int numParticles
)
{
    // compute the dimensions of the cuda grid for tgiven block dimensions,
    // and the number of particles

    if (numParticles % blockDimensions.x == 0)
    {
        gridDimensions.x = numParticles/blockDimensions.x;
    }
    else
    {
        gridDimensions.x = numParticles/blockDimensions.x + 1;    
    }
    gridDimensions.y = 1;
    gridDimensions.z = 1;
}
//------------------------------------------------------------------------------

//==============================================================================
// SPHParticleData's definition
//==============================================================================

//------------------------------------------------------------------------------
Solver::SPHParticleData::SPHParticleData (
    ParticleData* data, 
    unsigned int numGridCells   // # of grid cells in each direction
)
: 
    Data(data),
    BlockDimensions(BLOCK_DIMENSIONS_X, 1, 1), 
    NumGridCells(numGridCells)
{
    // allocate additional memory for storing density, pressure, acceleration,
    // velocities and hash values for the particles and initialize that data.
    // also allocate memory for the neighbor search as described in the nVidia
    // particles white paper
    CUDA::Alloc<float>(&dDensities, data->MaxParticles);
    CUDA::Alloc<float>(&dPressures, data->MaxParticles);
    CUDA::Alloc<float>(&dAccelerations, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dVelocities, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dTempPositions, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dTempVelocities, 3*data->MaxParticles);
    CUDA::Alloc<float>(&dBlendCoefficients, data->MaxParticles);
    CUDA::Alloc<float>(&dTempBlendCoefficients, data->MaxParticles);
    CUDA::Alloc<unsigned char>(&dStates, data->MaxParticles);
    CUDA::Alloc<unsigned char>(&dTempStates, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dNumParticles, 1);
    CUDA::Fill<float>(dDensities, data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dPressures, data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dAccelerations, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dVelocities, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dTempPositions, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dTempVelocities, 3*data->MaxParticles, 0.0f);
    CUDA::Fill<float>(dBlendCoefficients, data->MaxParticles, 1.0f);
    CUDA::Fill<float>(dTempBlendCoefficients, data->MaxParticles, 1.0f);
    CUDA::Fill<unsigned char>(dStates, data->MaxParticles, PS_DEFAULT);
    CUDA::Fill<unsigned char>(dTempStates, data->MaxParticles, PS_DEFAULT);
    CUDA::Memcpy<unsigned int>(
        dNumParticles, 
        &Data->NumParticles, 
        1, 
        hipMemcpyHostToDevice
    );
    CUDA::Alloc<unsigned int>(&dActiveIDs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dHashs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dCellStart, numGridCells);
    CUDA::Alloc<unsigned int>(&dCellEnd, numGridCells);
    CUDA::Fill<unsigned int>(dActiveIDs, data->MaxParticles, 0, 1);
    CUDA::Fill<unsigned int>(dHashs, data->MaxParticles, 0);
    CUDA::Fill<unsigned int>(dCellStart, numGridCells, 0);
    CUDA::Fill<unsigned int>(dCellEnd, numGridCells, 0);

    // compute the number of cuda blocks we need based on the current number
    // of particles and the threads per block we use. Also compute the amount
    // of shared memory we need to compute the values for [dCellStart] and
    // [dCellEnd]
    computeGridDimensions(GridDimensions, BlockDimensions, data->NumParticles);
    SharedMemSize = sizeof(int)*(BlockDimensions.x + 1);
}
//------------------------------------------------------------------------------
Solver::SPHParticleData::~SPHParticleData ()
{
    // free everything

    CUDA::Free<float>(&dDensities);
    CUDA::Free<float>(&dAccelerations);
    CUDA::Free<float>(&dPressures);
    CUDA::Free<float>(&dVelocities);
    CUDA::Free<float>(&dTempVelocities);
    CUDA::Free<float>(&dTempPositions);
    CUDA::Free<float>(&dBlendCoefficients);
    CUDA::Free<float>(&dTempBlendCoefficients);
    CUDA::Free<unsigned char>(&dStates);
    CUDA::Free<unsigned char>(&dTempStates);
    CUDA::Free<unsigned int>(&dHashs);
    CUDA::Free<unsigned int>(&dCellStart);
    CUDA::Free<unsigned int>(&dCellEnd);
}
//------------------------------------------------------------------------------
Solver::BoundaryParticleData::BoundaryParticleData (
    ParticleData* data, 
    unsigned int numGridCells
)
:
    Data(data),
    BlockDimensions(BLOCK_DIMENSIONS_X, 1, 1), 
    NumGridCells(numGridCells)
{
    CUDA::Alloc<unsigned int>(&dHashs, data->MaxParticles);
    CUDA::Alloc<unsigned int>(&dCellStart, numGridCells);
    CUDA::Alloc<unsigned int>(&dCellEnd, numGridCells);
    CUDA::Fill<unsigned int>(dHashs, data->MaxParticles, 0);
    CUDA::Fill<unsigned int>(dCellStart, numGridCells, 0);
    CUDA::Fill<unsigned int>(dCellEnd, numGridCells, 0);    

    computeGridDimensions(GridDimensions, BlockDimensions, data->NumParticles);
    SharedMemSize = sizeof(int)*(BlockDimensions.x + 1);
}
//------------------------------------------------------------------------------
Solver::BoundaryParticleData::~BoundaryParticleData ()
{
    CUDA::Free<unsigned int>(&dHashs);
    CUDA::Free<unsigned int>(&dCellStart);
    CUDA::Free<unsigned int>(&dCellEnd);
}
//------------------------------------------------------------------------------

//==============================================================================
// Solvers's definition
//==============================================================================

//------------------------------------------------------------------------------
Solver::Solver 
(
    ParticleData* fluidData, 
    ParticleData* fluidDataHigh,
    ParticleData* boundaryData,
    const SolverConfiguration* configuration
)
: 
    mConfiguration(*configuration), 
    mFluidData(
        fluidData, 
        Grid::ComputeNumGridCells(configuration->Grid[0])
    ), 
    mFluidDataHigh(
        fluidDataHigh, 
        Grid::ComputeNumGridCells(configuration->Grid[1])
    ), 
    mBoundaryData(
        boundaryData, 
        Grid::ComputeNumGridCells(configuration->Grid[1])
    )
{
    // store pointer to fluid particles and boundary particles also store
    // a copy of solver configuration

    //--------------------------------------------------------------------------
    // compute neighborhood of boundary particles beforehand 
    unsigned int* dBoundaryIDs;
    CUDA::Alloc<unsigned int>(&dBoundaryIDs, mBoundaryData.Data->MaxParticles);
    CUDA::Fill<unsigned int>(dBoundaryIDs, mBoundaryData.Data->MaxParticles, 
        0, 1);

    float* dBoundaryPositions;
    CUDA::Alloc<float>(&dBoundaryPositions, 3*mBoundaryData.Data->MaxParticles);
    
    this->Bind();   // bind first
    mBoundaryData.Data->Map();
    computeHashsLowD<<<mBoundaryData.GridDimensions, 
        mBoundaryData.BlockDimensions>>>(
        mBoundaryData.dHashs, 
        dBoundaryIDs, 
        mBoundaryData.Data->dPositions, 
        mBoundaryData.Data->NumParticles
    );
    thrust::sort_by_key(
        thrust::device_ptr<unsigned int>(mBoundaryData.dHashs),
        thrust::device_ptr<unsigned int>(mBoundaryData.dHashs + 
            mBoundaryData.Data->NumParticles),
        thrust::device_ptr<unsigned int>(dBoundaryIDs)
    );
    CUDA::Memset<unsigned int>(
        mBoundaryData.dCellStart, 
        EMPTY_CELL_ID, 
        mBoundaryData.NumGridCells
    );
    CUDA::Memset<unsigned int>(
        mBoundaryData.dCellEnd, 
        EMPTY_CELL_ID, 
        mBoundaryData.NumGridCells
    );
    reorderComputeCellStartEndBoundaryD<<<mBoundaryData.GridDimensions,
        mBoundaryData.BlockDimensions, 
        mBoundaryData.SharedMemSize>>>(
        mBoundaryData.dCellStart,
        mBoundaryData.dCellEnd,
        dBoundaryPositions,
        mBoundaryData.Data->dPositions,
        dBoundaryIDs,
        mBoundaryData.dHashs,
        mBoundaryData.Data->NumParticles 
    );
    CUDA::Memcpy<float>(
        mBoundaryData.Data->dPositions, 
        dBoundaryPositions, 
        3*mBoundaryData.Data->MaxParticles,
        hipMemcpyDeviceToDevice
    );
    mBoundaryData.Data->Unmap();

    CUDA::Free<unsigned int>(&dBoundaryIDs);
    CUDA::Free<float>(&dBoundaryPositions);
    //--------------------------------------------------------------------------

}
//------------------------------------------------------------------------------
Solver::~Solver ()
{

}
//------------------------------------------------------------------------------
void Solver::Bind () const
{
    // set the configuration of this solver on the device
    CUDA::SafeCall(
        hipMemcpyToSymbol(HIP_SYMBOL(
            gConfiguration), 
            &mConfiguration, 
            sizeof(mConfiguration)
        ), 
        __FILE__, 
        __LINE__ 
    );
}
//------------------------------------------------------------------------------
void Solver::Advance (float timeStep)
{
    CUDA::Timer t;
    t.Start();
    mFluidData.Data->Map();
    mFluidDataHigh.Data->Map();
    mBoundaryData.Data->Map();
    this->computeNeighborhoodsLow();
    this->computeNeighborhoodsHigh();
    this->computeDensitiesLow();
    this->computeDensitiesHigh();
    //CUDA::DumpArray<float>(mFluidDataHigh.dDensities, mFluidData.Data->NumParticles);
    this->computeAccelerationsAndUpdateStatesLow();
    this->computeAccelerationsAndUpdateStatesHigh();
    this->updateSystem(mFluidData, 0.0f);
    this->updateSystem(mFluidDataHigh, timeStep);
    mBoundaryData.Data->Unmap();
    mFluidDataHigh.Data->Unmap();
    mFluidData.Data->Unmap();
    t.Stop();
    t.DumpElapsed();
}
//------------------------------------------------------------------------------
void Solver::computeNeighborhoodsLow ()
{    
    // copy back from device the new amount of particles for this time step
    CUDA::Memcpy<unsigned int>(
        &mFluidData.Data->NumParticles,
        mFluidData.dNumParticles,
        1,
        hipMemcpyDeviceToHost
    );

    // compute hashs of all particles
    computeHashsLowD<<<mFluidData.GridDimensions, mFluidData.BlockDimensions>>>(
        mFluidData.dHashs,
        mFluidData.dActiveIDs,
        mFluidData.Data->dPositions,
        mFluidData.Data->NumParticles
    );
        
    // sort the active particle ids by their hash
    thrust::sort_by_key(
        thrust::device_ptr<unsigned int>(mFluidData.dHashs),
        thrust::device_ptr<unsigned int>(mFluidData.dHashs + 
            mFluidData.Data->NumParticles),
        thrust::device_ptr<unsigned int>(mFluidData.dActiveIDs)
    );

    // reset, then compute cell start end list
    CUDA::Memset<unsigned int>(
        mFluidData.dCellStart, 
        EMPTY_CELL_ID, 
        mFluidData.NumGridCells
    );
    CUDA::Memset<unsigned int>(
        mFluidData.dCellEnd, 
        EMPTY_CELL_ID, 
        mFluidData.NumGridCells
    );
    reorderAndComputeCellStartEndD<<<mFluidData.GridDimensions, 
        mFluidData.BlockDimensions, mFluidData.SharedMemSize>>>(
        mFluidData.dCellStart,
        mFluidData.dCellEnd,
        mFluidData.dTempPositions,
        mFluidData.dTempVelocities,
        mFluidData.dTempBlendCoefficients,
        mFluidData.dTempStates,
        mFluidData.dActiveIDs,
        mFluidData.Data->dPositions,
        mFluidData.dVelocities,
        mFluidData.dBlendCoefficients,
        mFluidData.dStates,
        mFluidData.dHashs,
        mFluidData.Data->NumParticles
    );

}
//------------------------------------------------------------------------------
void Solver::computeNeighborhoodsHigh ()
{
    // copy back from device the new amount of particles for this time step
    CUDA::Memcpy<unsigned int>(
        &mFluidDataHigh.Data->NumParticles,
        mFluidDataHigh.dNumParticles,
        1,
        hipMemcpyDeviceToHost
    );

    // compute hashs of all particles
    computeHashsHighD<<<mFluidDataHigh.GridDimensions, 
        mFluidDataHigh.BlockDimensions>>>(
        mFluidDataHigh.dHashs,
        mFluidDataHigh.dActiveIDs,
        mFluidDataHigh.Data->dPositions,
        mFluidDataHigh.Data->NumParticles
    );
        
    // sort the active particle ids by their hash
    thrust::sort_by_key(
        thrust::device_ptr<unsigned int>(mFluidDataHigh.dHashs),
        thrust::device_ptr<unsigned int>(mFluidDataHigh.dHashs + 
            mFluidDataHigh.Data->NumParticles),
        thrust::device_ptr<unsigned int>(mFluidDataHigh.dActiveIDs)
    );

    // reset, then compute cell start end list
    CUDA::Memset<unsigned int>(
        mFluidDataHigh.dCellStart, 
        EMPTY_CELL_ID, 
        mFluidDataHigh.NumGridCells
    );
    CUDA::Memset<unsigned int>(
        mFluidDataHigh.dCellEnd, 
        EMPTY_CELL_ID, 
        mFluidDataHigh.NumGridCells
    );
    reorderAndComputeCellStartEndD<<<mFluidDataHigh.GridDimensions, 
        mFluidDataHigh.BlockDimensions, mFluidDataHigh.SharedMemSize>>>(
        mFluidDataHigh.dCellStart,
        mFluidDataHigh.dCellEnd,
        mFluidDataHigh.dTempPositions,
        mFluidDataHigh.dTempVelocities,
        mFluidDataHigh.dTempBlendCoefficients,
        mFluidDataHigh.dTempStates,
        mFluidDataHigh.dActiveIDs,
        mFluidDataHigh.Data->dPositions,
        mFluidDataHigh.dVelocities,
        mFluidDataHigh.dBlendCoefficients,
        mFluidDataHigh.dStates,
        mFluidDataHigh.dHashs,
        mFluidDataHigh.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
void Solver::computeDensitiesLow ()
{
    computeDensitiesPressuresLowD<<<mFluidData.GridDimensions, 
        mFluidData.BlockDimensions>>>(
        mFluidData.dDensities,
        mFluidData.dPressures,
        mFluidData.dTempPositions,
        mFluidDataHigh.dTempPositions,
        mFluidData.dCellStart,
        mFluidData.dCellEnd,
        mFluidDataHigh.dCellStart,
        mFluidDataHigh.dCellEnd,
        mFluidData.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
void Solver::computeDensitiesHigh ()
{
    computeDensitiesPressuresHighD<<<mFluidDataHigh.GridDimensions, 
        mFluidDataHigh.BlockDimensions>>>(
        mFluidDataHigh.dDensities,
        mFluidDataHigh.dPressures,
        mFluidDataHigh.dTempPositions,
        mFluidData.dTempPositions,
        mFluidDataHigh.dCellStart,
        mFluidDataHigh.dCellEnd,
        mFluidData.dCellStart,
        mFluidData.dCellEnd,
        mFluidDataHigh.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
void Solver::computeAccelerationsAndUpdateStatesLow ()
{
    // computes the accelerations

    computeAccelerationsAndUpdateStatesLowD<<<mFluidData.GridDimensions, 
        mFluidData.BlockDimensions>>>(
        mFluidData.dAccelerations,
        mFluidData.dTempStates,
        mFluidData.dDensities,
        mFluidData.dPressures,
        mFluidData.dTempPositions,
        mFluidData.dTempVelocities,
        mFluidDataHigh.dDensities,
        mFluidDataHigh.dPressures,
        mFluidDataHigh.dTempPositions,
        mFluidDataHigh.dTempVelocities,
        mFluidData.dCellStart,
        mFluidData.dCellEnd,
        mFluidDataHigh.dCellStart,
        mFluidDataHigh.dCellEnd,
        mBoundaryData.Data->dPositions,
        mBoundaryData.dCellStart,
        mBoundaryData.dCellEnd,
        mFluidData.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
void Solver::computeAccelerationsAndUpdateStatesHigh ()
{
    computeAccelerationsAndUpdateStatesHighD<<<mFluidDataHigh.GridDimensions, 
        mFluidDataHigh.BlockDimensions>>>(
        mFluidDataHigh.dAccelerations,
        mFluidDataHigh.dTempStates,
        mFluidDataHigh.dDensities,
        mFluidDataHigh.dPressures,
        mFluidDataHigh.dTempPositions,
        mFluidDataHigh.dTempVelocities,
        mFluidData.dDensities,
        mFluidData.dPressures,
        mFluidData.dTempPositions,
        mFluidData.dTempVelocities,
        mFluidDataHigh.dCellStart,
        mFluidDataHigh.dCellEnd,
        mFluidData.dCellStart,
        mFluidData.dCellEnd,
        mBoundaryData.Data->dPositions,
        mBoundaryData.dCellStart,
        mBoundaryData.dCellEnd,
        mFluidDataHigh.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
void Solver::updateSystem (SPHParticleData& fluidData, float timeStep)
{
    // update:
    //  - active particles positions and velocities
    //  - active particles blendvalues according to their state
    //  - the IDs list (i.e. don't add (remove) particles, that have a 
    //    blend values of zero)


    // first set the number of particles to zero, as the following CUDA kernel
    // may delete particles from the system and counts the new amount of 
    // particles 
    CUDA::Memset<unsigned int>(fluidData.dNumParticles, 0, 1);

    updateSystemD<<<fluidData.GridDimensions, 
        fluidData.BlockDimensions>>>(
        fluidData.Data->dPositions,
        fluidData.dVelocities,
        fluidData.dBlendCoefficients,
        fluidData.dStates,
        fluidData.dActiveIDs,
        fluidData.dNumParticles,
        fluidData.dTempPositions,
        fluidData.dTempVelocities,
        fluidData.dTempBlendCoefficients,
        fluidData.dTempStates,
        fluidData.dAccelerations,
        timeStep,
        fluidData.Data->NumParticles
    );
}
//------------------------------------------------------------------------------
